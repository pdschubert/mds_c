#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2015 - 2016 Philipp Schubert.                                *
 * All rights reserved. This program and the accompanying materials are made  *
 * available under the terms of LICENSE.txt.                                  *
 *                                                                            *
 * Contributors:                                                              *
 *     Philipp Schubert                                                       *
 *****************************************************************************/

/** @file cuda_disfuncs.cu
 *  @brief Implementation der Prototypen aus cuda_disfuncs.cuh.
 *
 *  Hier sind die Wrapperfunktionen der Prototypen aus cuda_disfuncs.cuh
 *  implementiert, sowie entsprechende CUDA Kernel, die die Berechnungen auf der
 *  CUDA Karte durchführt.
 *
 *  @author Philipp D. Schubert
 *  @bug Keine Bugs bekannt.
 */

#include <stdio.h>
#include <stdlib.h>

/**
 * @brief Zu benutzende Größe der TILES im den Kernels dieses Moduls.
 */
#define TILE_WIDTH 16

// my C includes
extern "C" {
#include "m.h"
#include "tm.h"
#include "utils.cuh"
#include "utils.h"
}

/**
 * @brief Test Kernel zur Überprüfung des Moduls cuda_disfuncs.cu.
 */
__global__ void kernel_disfuncs_test() {
  printf("cuda_disfuncs: working fine!\n");
}

extern "C" void cuda_disfuncs_test() {
  kernel_disfuncs_test<<<1, 1>>>();
  hipDeviceSynchronize();
}

/**
 * @brief Berechnet aus gegebener Datenmatrix eine Distanzmatrix der
 * euklidischen Distanzen.
 *
 * Dieser Kernel berechnet aus gegebener Datenmatrix einer Matrix, die die
 * euklidischen Distanzen enthält. Zur Berechnung dieser Distanzmatrix wird eine
 * spezielle Variante der allgemeinen Matrixmultiplikation aus cuda_linalg.cu
 * verwendet. Der geteilte Speicher At und Bt wird dazu ausgenutzt. At wird mit
 * Daten aus der Matrix a befüllt, Bt mit den transponierten Daten aus der
 * Matrix a. Auf diese weise kann die Distanzmatrix sehr geschickt berechnet
 * werden, wobei die Graphikkarte nahezu optimal ausgelastet wird.
 *
 * @warning GPU Kernel Funktion
 * @param a Datenmatrix
 * @param arows Anzahl der Zeilen der Datenmatrix
 * @param acols Anzahl der Spalten der Datenmatrix
 * @param splen Anzahl der mindestens benötigten Blöcke, um die Matrix in
 * Dimension 1 abzudecken
 * @param dist untere Dreiecksmatrix in die die Distanzen geschrieben werden
 */
__global__ void kernel_distanceMatrix(const real_t *a, const unsigned int arows,
                                      const unsigned int acols,
                                      const unsigned int splen, real_t *dist) {
  __shared__ real_t At[TILE_WIDTH][TILE_WIDTH];
  __shared__ real_t Bt[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int prow = bx * TILE_WIDTH + ty;
  int pcol = by * TILE_WIDTH + tx;
  real_t dist_tmp = 0.0;
  real_t diff;
  for (int m = 0; m < splen; ++m) {
    /* int aax = prow; */
    int aay = m * TILE_WIDTH + tx;
    int bbx = m * TILE_WIDTH + ty;
    /* int bby = pcol; */
    At[ty][tx] = (prow < arows && aay < acols) ? a[prow * acols + aay] : 0.0;
    Bt[ty][tx] = (bbx < acols && pcol < arows) ? a[pcol * acols + bbx] : 0.0;
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      diff = (At[ty][k] - Bt[k][tx]);
      dist_tmp += diff * diff;
    }
    __syncthreads();
    if (prow < arows && pcol < arows && (pcol <= prow))
      dist[(prow * (prow + 1)) / 2 + pcol] = sqrtf(dist_tmp);
  }
}

extern "C" tm_t *cuda_distanceMatrix(const m_t *data) {
  tm_t *dist = initTM(data->rows);
  real_t *dev_data, *dev_dist;
  CUERROR(hipMalloc((void **)&dev_data, data->num_elems * sizeof(real_t)));
  CUERROR(hipMalloc((void **)&dev_dist, dist->num_elems * sizeof(real_t)));
  CUERROR(hipMemcpy(dev_data, data->elems, data->num_elems * sizeof(real_t),
                     hipMemcpyHostToDevice));
  int gdim1, gdim2;
  gdim1 = gdim2 = (data->rows % TILE_WIDTH == 0)
                      ? data->rows / TILE_WIDTH
                      : (data->rows / TILE_WIDTH) + 1;
  int scalprodlen = (data->cols % TILE_WIDTH == 0)
                        ? data->cols / TILE_WIDTH
                        : (data->cols / TILE_WIDTH) + 1;
  dim3 gridsize(gdim1, gdim2);
  dim3 blocksize(TILE_WIDTH, TILE_WIDTH);
  kernel_distanceMatrix<<<gridsize, blocksize>>>(
      dev_data, data->rows, data->cols, scalprodlen, dev_dist);
  CUERROR(hipMemcpy(dist->elems, dev_dist, dist->num_elems * sizeof(real_t),
                     hipMemcpyDeviceToHost));
  hipFree(dev_data);
  hipFree(dev_dist);
  return dist;
}

extern "C" void cuda_distanceMatrix_nomem(const real_t *dev_x,
                                          unsigned int xrows,
                                          unsigned int xcols, real_t *dev_d) {
  int gdim1, gdim2;
  gdim1 = gdim2 =
      (xrows % TILE_WIDTH == 0) ? xrows / TILE_WIDTH : (xrows / TILE_WIDTH) + 1;
  int scalprodlen =
      (xcols % TILE_WIDTH == 0) ? xcols / TILE_WIDTH : (xcols / TILE_WIDTH) + 1;
  dim3 gridsize(gdim1, gdim2);
  dim3 blocksize(TILE_WIDTH, TILE_WIDTH);
  kernel_distanceMatrix<<<gridsize, blocksize>>>(dev_x, xrows, xcols,
                                                 scalprodlen, dev_d);
}
